#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include <cmath>

#include "./math_functions.hpp"

namespace caffe {

void caffe_gpu_gemm(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

void caffe_gpu_gemv(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float* A, const float* x,
    const float beta, float* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasSgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

void caffe_gpu_axpy(const int N, const float alpha, const float* X, float* Y) {
  CUBLAS_CHECK(hipblasSaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

void caffe_gpu_memcpy(const size_t N, const void* X, void* Y) {
  if (X != Y) {
    CUDA_CHECK(hipMemcpy(Y, X, N, hipMemcpyDefault));  // NOLINT(caffe/alt_fn)
  }
}

void caffe_gpu_scal(const int N, const float alpha, float *X) {
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

void caffe_gpu_axpby(const int N, const float alpha, const float* X,
                     const float beta, float* Y) {
  caffe_gpu_scal(N, beta, Y);
  caffe_gpu_axpy(N, alpha, X, Y);
}

void caffe_gpu_dot(const int n, const float* x, const float* y, float* out) {
  CUBLAS_CHECK(hipblasSdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

void caffe_gpu_asum(const int n, const float* x, float* y) {
  CUBLAS_CHECK(hipblasSasum(Caffe::cublas_handle(), n, x, 1, y));
}

void caffe_gpu_scale(const int n, const float alpha, const float *x,
                     float* y) {
  CUBLAS_CHECK(hipblasScopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

__global__ void set_kernel(const int n, const real_t alpha, real_t* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = alpha;
  }
}

void caffe_gpu_set(const int N, const real_t alpha, real_t* Y) {
  if (alpha == 0) {
    CUDA_CHECK(hipMemset(Y, 0, sizeof(real_t) * N));  // NOLINT(caffe/alt_fn)
    return;
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  set_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

__global__ void add_scalar_kernel(const int n, const real_t alpha, real_t* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] += alpha;
  }
}

void caffe_gpu_add_scalar(const int N, const float alpha, float* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

__global__ void add_kernel(const int n, const real_t* a,
    const real_t* b, real_t* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] + b[index];
  }
}

void caffe_gpu_add(const int N, const float* a, const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

__global__ void sub_kernel(const int n, const real_t* a,
    const real_t* b, real_t* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] - b[index];
  }
}

void caffe_gpu_sub(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

__global__ void mul_kernel(const int n, const real_t* a,
    const real_t* b, real_t* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] * b[index];
  }
}

void caffe_gpu_mul(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

__global__ void div_kernel(const int n, const real_t* a,
    const real_t* b, real_t* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] / b[index];
  }
}

void caffe_gpu_div(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

__global__ void abs_kernel(const int n, const real_t* a, real_t* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = abs(a[index]);
  }
}

void caffe_gpu_abs(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

__global__ void exp_kernel(const int n, const real_t* a, real_t* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = exp(a[index]);
  }
}

void caffe_gpu_exp(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

__global__ void log_kernel(const int n, const real_t* a, real_t* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = log(a[index]);
  }
}

void caffe_gpu_log(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

__global__ void powx_kernel(const int n, const real_t* a,
    const real_t alpha, real_t* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = pow(a[index], alpha);
  }
}

void caffe_gpu_powx(const int N, const float* a,
    const float alpha, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

}  // namespace caffe
