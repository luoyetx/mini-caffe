#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "./permute_layer.hpp"
#include "../util/math_functions.hpp"

namespace caffe {

__global__ void PermuteKernel(const int nthreads,
    real_t* const bottom_data, const bool forward, const int* permute_order,
    const int* old_steps, const int* new_steps, const int num_axes,
    real_t* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int temp_idx = index;
    int old_idx = 0;
    for (int i = 0; i < num_axes; ++i) {
      int order = permute_order[i];
      old_idx += (temp_idx / new_steps[i]) * old_steps[order];
      temp_idx %= new_steps[i];
    }
    if (forward) {
      top_data[index] = bottom_data[old_idx];
    } else {
      bottom_data[old_idx] = top_data[index];
    }
  }
}

void PermuteLayer::Forward_gpu(const vector<Blob*>& bottom,
                               const vector<Blob*>& top) {
  if (need_permute_) {
    real_t* bottom_data = bottom[0]->mutable_gpu_data();
    real_t* top_data = top[0]->mutable_gpu_data();
    int count = top[0]->count();
    const int* permute_order = permute_order_.gpu_data();
    const int* new_steps = new_steps_.gpu_data();
    const int* old_steps = old_steps_.gpu_data();
    bool foward = true;
    // NOLINT_NEXT_LINE(whitespace/operators)
    PermuteKernel<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, foward, permute_order, old_steps, new_steps,
        num_axes_, top_data);
    CUDA_POST_KERNEL_CHECK;
  } else {
    // If there is no need to permute
    caffe_copy(bottom[0]->count(), bottom[0]->gpu_data(), top[0]->mutable_gpu_data());
  }
}

}  // namespace caffe
