#include "hip/hip_runtime.h"
#include <vector>
#include "./conv_dw_layer.hpp"
//#include "caffe/util/gpu_util.cuh"

namespace caffe {

template <typename Dtype>
__global__ void ConvolutionDepthwiseWeightForward(const int nthreads,
    const Dtype* const bottom_data, const Dtype* const weight_data, const int num, const int channels,
    const int top_height, const int top_width, const int bottom_height, const int bottom_width,
    const int kernel_h, const int kernel_w, const int stride_h, const int stride_w,
    const int pad_h, const int pad_w, const int dilation_h, const int dilation_w,
    Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / channels / top_height / top_width;
    const int c = (index / top_height / top_width) % channels;
    const int h = (index / top_width) % top_height;
    const int w = index % top_width;
    const Dtype* weight = weight_data + c * kernel_h * kernel_w;
    Dtype value = 0;
    for (int kh = 0; kh < kernel_h; ++kh)
    {
      for (int kw = 0; kw < kernel_w; ++kw)
      {
        const int h_in = -pad_h + h * stride_h + kh * dilation_h;
        const int w_in = -pad_w + w * stride_w + kw * dilation_w;
        if ((h_in >= 0) && (h_in < bottom_height) && (w_in >= 0) && (w_in < bottom_width))
        {
          const int offset = ((n * channels + c) * bottom_height + h_in) * bottom_width + w_in;
          value += (*weight) * bottom_data[offset];
        }
        ++weight;
      }
    }
    top_data[index] = value;
  }
}

template <typename Dtype>
__global__ void ConvolutionDepthwiseBiasForward(const int nthreads,
    const Dtype* const bias_data, const int num, const int channels,
    const int top_height, const int top_width, Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int c = (index / top_height / top_width) % channels;
    top_data[index] += bias_data[c];
  }
}

void ConvolutionDepthwiseLayer::Forward_gpu(const vector<Blob*>& bottom,
                                            const vector<Blob*>& top) {
  const real_t* bottom_data = bottom[0]->gpu_data();
  real_t* top_data = top[0]->mutable_gpu_data();
  const real_t* weight_data = this->blobs_[0]->gpu_data();
  const int count = top[0]->count();
  const int num = top[0]->num();
  const int channels = top[0]->channels();
  const int top_height = top[0]->height();
  const int top_width = top[0]->width();
  const int bottom_height = bottom[0]->height();
  const int bottom_width = bottom[0]->width();
  ConvolutionDepthwiseWeightForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, weight_data, num, channels,
      top_height, top_width, bottom_height, bottom_width,
      kernel_h_, kernel_w_, stride_h_, stride_w_,
      pad_h_, pad_w_, dilation_h_, dilation_w_, top_data);
  if (this->layer_param_.convolution_param().bias_term()) {
    const real_t* bias_data = this->blobs_[1]->gpu_data();
    ConvolutionDepthwiseBiasForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bias_data, num, channels,
        top_height, top_width, top_data);
  }
}

}  // namespace caffe
