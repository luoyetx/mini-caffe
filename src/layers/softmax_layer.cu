#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "./softmax_layer.hpp"
#include "../util/math_functions.hpp"

namespace caffe {

__global__ void kernel_channel_max(const int num, const int channels,
    const int spatial_dim, const real_t* data, real_t* out) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int n = index / spatial_dim;
    int s = index % spatial_dim;
    real_t maxval = -FLT_MAX;
    for (int c = 0; c < channels; ++c) {
      maxval = max(data[(n * channels + c) * spatial_dim + s], maxval);
    }
    out[index] = maxval;
  }
}

__global__ void kernel_channel_subtract(const int count,
    const int num, const int channels,
    const int spatial_dim, const real_t* channel_max, real_t* data) {
  CUDA_KERNEL_LOOP(index, count) {
    int n = index / channels / spatial_dim;
    int s = index % spatial_dim;
    data[index] -= channel_max[n * spatial_dim + s];
  }
}

__global__ void kernel_exp(const int count, const real_t* data, real_t* out) {
  CUDA_KERNEL_LOOP(index, count) {
    out[index] = exp(data[index]);
  }
}

__global__ void kernel_channel_sum(const int num, const int channels,
    const int spatial_dim, const real_t* data, real_t* channel_sum) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int n = index / spatial_dim;
    int s = index % spatial_dim;
    real_t sum = 0;
    for (int c = 0; c < channels; ++c) {
      sum += data[(n * channels + c) * spatial_dim + s];
    }
    channel_sum[index] = sum;
  }
}

__global__ void kernel_channel_div(const int count,
    const int num, const int channels,
    const int spatial_dim, const real_t* channel_sum, real_t* data) {
  CUDA_KERNEL_LOOP(index, count) {
    int n = index / channels / spatial_dim;
    int s = index % spatial_dim;
    data[index] /= channel_sum[n * spatial_dim + s];
  }
}

__global__ void kernel_channel_dot(const int num, const int channels,
    const int spatial_dim, const real_t* data_1, const real_t* data_2,
    real_t* channel_dot) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int n = index / spatial_dim;
    int s = index % spatial_dim;
    real_t dot = 0;
    for (int c = 0; c < channels; ++c) {
      dot += (data_1[(n * channels + c) * spatial_dim + s]
          * data_2[(n * channels + c) * spatial_dim + s]);
    }
    channel_dot[index] = dot;
  }
}

void SoftmaxLayer::Forward_gpu(const vector<Blob*>& bottom,
                               const vector<Blob*>& top) {
  const real_t* bottom_data = bottom[0]->gpu_data();
  real_t* top_data = top[0]->mutable_gpu_data();
  real_t* scale_data = scale_.mutable_gpu_data();
  int count = bottom[0]->count();
  int channels = top[0]->shape(softmax_axis_);
  caffe_copy(count, bottom_data, top_data);
  // We need to subtract the max to avoid numerical issues, compute the exp,
  // and then normalize.
  // compute max
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_max<<<CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
      CAFFE_CUDA_NUM_THREADS>>>(outer_num_, channels, inner_num_, top_data,
      scale_data);
  // subtract
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_subtract<<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS>>>(count, outer_num_, channels, inner_num_,
      scale_data, top_data);
  // exponentiate
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_exp<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, top_data, top_data);
  // sum after exp
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_sum<<<CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
      CAFFE_CUDA_NUM_THREADS>>>(outer_num_, channels, inner_num_, top_data,
      scale_data);
  // divide
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_div<<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS>>>(count, outer_num_, channels, inner_num_,
      scale_data, top_data);
}

}  // namespace caffe
