
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/filler.hpp"
#include "caffe/layers/cudnn_bn_layer.hpp"

#if CUDNN_VERSION_MIN(4, 0, 0)

namespace caffe {

template <typename Dtype>
void CuDNNBNLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const Dtype* scale_data = this->blobs_[0]->gpu_data();
  const Dtype* bias_data = this->blobs_[1]->gpu_data();

  if (this->phase_ == TEST) {
    const Dtype* running_mean_data = this->blobs_[2]->gpu_data();
    const Dtype* running_inv_variance_data = this->blobs_[3]->gpu_data();
    CUDNN_CHECK(cudnnBatchNormalizationForwardInference(handle_,
        CUDNN_BATCHNORM_SPATIAL,
        cudnn::dataType<Dtype>::one,
        cudnn::dataType<Dtype>::zero,
        bottom_desc_, bottom_data,
        top_desc_, top_data,
        bn_param_desc_, scale_data, bias_data,
        running_mean_data, running_inv_variance_data,
        this->bn_eps_));
  } else {
    Dtype* running_mean_data = this->blobs_[2]->mutable_gpu_data();
    Dtype* running_inv_variance_data = this->blobs_[3]->mutable_gpu_data();
    Dtype* save_mean_data = save_mean_.mutable_gpu_data();
    Dtype* save_inv_variance_data = save_inv_variance_.mutable_gpu_data();
    CUDNN_CHECK(cudnnBatchNormalizationForwardTraining(handle_,
        CUDNN_BATCHNORM_SPATIAL,
        cudnn::dataType<Dtype>::one,
        cudnn::dataType<Dtype>::zero,
        bottom_desc_, bottom_data,
        top_desc_, top_data,
        bn_param_desc_, scale_data, bias_data,
        this->bn_momentum_,
        running_mean_data, running_inv_variance_data,
        this->bn_eps_,
        save_mean_data, save_inv_variance_data));
  }
}

template <typename Dtype>
void CuDNNBNLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0] || this->param_propagate_down_[0] ||
      this->param_propagate_down_[1]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* scale_data = this->blobs_[0]->gpu_data();
    Dtype* scale_diff = scale_buf_.mutable_gpu_diff();
    Dtype* bias_diff = bias_buf_.mutable_gpu_diff();
    const Dtype* save_mean_data = save_mean_.gpu_data();
    const Dtype* save_inv_variance_data = save_inv_variance_.gpu_data();

    CUDNN_CHECK(cudnnBatchNormalizationBackward(handle_,
        CUDNN_BATCHNORM_SPATIAL,
        cudnn::dataType<Dtype>::one,
        cudnn::dataType<Dtype>::zero,
        bottom_desc_, bottom_data,
        top_desc_, top_diff,
        bottom_desc_, bottom_diff,
        bn_param_desc_, scale_data,
        scale_diff, bias_diff,
        this->bn_eps_,
        save_mean_data, save_inv_variance_data));

    if (this->param_propagate_down_[0]) {
      caffe_gpu_add(scale_buf_.count(), scale_diff,
          this->blobs_[0]->gpu_diff(), this->blobs_[0]->mutable_gpu_diff());
    }
    if (this->param_propagate_down_[1]) {
      caffe_gpu_add(bias_buf_.count(), bias_diff,
          this->blobs_[1]->gpu_diff(), this->blobs_[1]->mutable_gpu_diff());
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNBNLayer);

}  // namespace caffe
#endif
#endif
