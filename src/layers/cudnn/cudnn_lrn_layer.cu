
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN

#include "./cudnn_lrn_layer.hpp"

namespace caffe {

void CuDNNLRNLayer::Forward_gpu(const vector<Blob*>& bottom,
                                const vector<Blob*>& top) {
  const real_t* bottom_data = bottom[0]->gpu_data();
  real_t* top_data = top[0]->mutable_gpu_data();

  CUDNN_CHECK(cudnnLRNCrossChannelForward(
              handle_, norm_desc_, CUDNN_LRN_CROSS_CHANNEL_DIM1,
              cudnn::dataType<real_t>::one,
              bottom_desc_, bottom_data,
              cudnn::dataType<real_t>::zero,
              top_desc_, top_data));
}

};  // namespace caffe

#endif  // USE_CUDNN
