#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "../filler.hpp"
#include "./normalize_layer.hpp"
#include "../util/math_functions.hpp"

namespace caffe {

// divid a matrix with vector
template <typename Dtype>
__global__ void DivBsx(const int nthreads, const Dtype* A,
    const Dtype* v, const int rows, const int cols, const CBLAS_TRANSPOSE trans,
    Dtype* B) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int c = index % cols;
    int r = (index / cols) % rows;
    if (trans == CblasNoTrans) {
      B[index] = A[index] / v[c];
    } else {
      B[index] = A[index] / v[r];
    }
  }
}

template <typename Dtype>
__global__ void MulBsx(const int nthreads, const Dtype* A,
    const Dtype* v, const int rows, const int cols, const CBLAS_TRANSPOSE trans,
    Dtype* B) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int c = index % cols;
    int r = (index / cols) % rows;
    if (trans == CblasNoTrans) {
      B[index] = A[index] * v[c];
    } else {
      B[index] = A[index] * v[r];
    }
  }
}

void NormalizeLayer::Forward_gpu(const vector<Blob*>& bottom,
                                 const vector<Blob*>& top) {
  const real_t* bottom_data = bottom[0]->gpu_data();
  real_t* top_data = top[0]->mutable_gpu_data();
  real_t* buffer_data = buffer_.mutable_gpu_data();
  real_t* norm_data;
  if (across_spatial_) {
    // need to index it
    norm_data = norm_.mutable_cpu_data();
  } else {
    norm_data = norm_.mutable_gpu_data();
    // add eps to avoid overflow
    caffe_gpu_set(norm_.count(), real_t(eps_), norm_data);
  }
  const real_t* scale;
  if (channel_shared_) {
    scale = this->blobs_[0]->cpu_data();
  } else {
    scale = this->blobs_[0]->gpu_data();
  }
  const real_t* sum_channel_multiplier = sum_channel_multiplier_.gpu_data();
  int num = bottom[0]->num();
  int dim = bottom[0]->count() / num;
  int spatial_dim = bottom[0]->height() * bottom[0]->width();
  int channels = bottom[0]->channels();
  for (int n = 0; n < num; ++n) {
    caffe_gpu_powx(dim, bottom_data, real_t(2), buffer_data);
    if (across_spatial_) {
      real_t normsqr;
      caffe_gpu_asum(dim, buffer_data, &normsqr);
      // add eps to avoid overflow
      norm_data[n] = pow(normsqr+eps_, real_t(0.5));
      caffe_gpu_scale(dim, real_t(1.0 / norm_data[n]), bottom_data, top_data);
    } else {
      // compute norm
      caffe_gpu_gemv(CblasTrans, channels, spatial_dim, real_t(1),
                            buffer_data, sum_channel_multiplier, real_t(1),
                            norm_data);
      caffe_gpu_powx(spatial_dim, norm_data, real_t(0.5), norm_data);
      // scale the layer
      // NOLINT_NEXT_LINE(whitespace/operators)
      DivBsx<<<CAFFE_GET_BLOCKS(dim), CAFFE_CUDA_NUM_THREADS>>>(
          dim, bottom_data, norm_data, channels, spatial_dim, CblasNoTrans,
          top_data);
      CUDA_POST_KERNEL_CHECK;
      norm_data += spatial_dim;
    }
    // scale the output
    if (channel_shared_) {
      caffe_gpu_scal(dim, scale[0], top_data);
    } else {
      // NOLINT_NEXT_LINE(whitespace/operators)
      MulBsx<<<CAFFE_GET_BLOCKS(dim), CAFFE_CUDA_NUM_THREADS>>>(
          dim, top_data, scale, channels, spatial_dim, CblasTrans,
          top_data);
      CUDA_POST_KERNEL_CHECK;
    }
    bottom_data += dim;
    top_data += dim;
  }
}

}  // namespace caffe
