#include "hip/hip_runtime.h"
#include <vector>

#include "./slice_layer.hpp"
#include "../util/math_functions.hpp"

namespace caffe {

__global__ void Slice(const int nthreads, const real_t* in_data,
    const bool forward, const int num_slices, const int slice_size,
    const int bottom_slice_axis, const int top_slice_axis,
    const int offset_slice_axis, real_t* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int total_slice_size = slice_size * top_slice_axis;
    const int slice_num = index / total_slice_size;
    const int slice_index = index % total_slice_size;
    const int bottom_index = slice_index +
        (slice_num * bottom_slice_axis + offset_slice_axis) * slice_size;
    if (forward) {
      out_data[index] = in_data[bottom_index];
    } else {
      out_data[bottom_index] = in_data[index];
    }
  }
}

void SliceLayer::Forward_gpu(const vector<Blob*>& bottom,
                             const vector<Blob*>& top) {
  if (top.size() == 1) { return; }
  int offset_slice_axis = 0;
  const real_t* bottom_data = bottom[0]->gpu_data();
  const int bottom_slice_axis = bottom[0]->shape(slice_axis_);
  const bool kForward = true;
  for (int i = 0; i < top.size(); ++i) {
    real_t* top_data = top[i]->mutable_gpu_data();
    const int top_slice_axis = top[i]->shape(slice_axis_);
    const int top_slice_size = top_slice_axis * slice_size_;
    const int nthreads = top_slice_size * num_slices_;
    Slice // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, bottom_data, kForward, num_slices_, slice_size_,
        bottom_slice_axis, top_slice_axis, offset_slice_axis, top_data);
    offset_slice_axis += top_slice_axis;
  }
}

}  // namespace caffe
