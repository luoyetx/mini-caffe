#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>

#include "./sigmoid_layer.hpp"

namespace caffe {

__global__ void SigmoidForward(const int n, const real_t* in, real_t* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = 1. / (1. + exp(-in[index]));
  }
}

void SigmoidLayer::Forward_gpu(const vector<Blob*>& bottom,
                               const vector<Blob*>& top) {
  const real_t* bottom_data = bottom[0]->gpu_data();
  real_t* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  SigmoidForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

}  // namespace caffe
