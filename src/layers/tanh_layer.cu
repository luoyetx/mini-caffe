#include "hip/hip_runtime.h"
// TanH neuron activation function layer.
// Adapted from ReLU layer code written by Yangqing Jia

#include <vector>

#include "./tanh_layer.hpp"

namespace caffe {

__global__ void TanHForward(const int n, const real_t* in, real_t* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = tanh(in[index]);
  }
}

void TanHLayer::Forward_gpu(const vector<Blob*>& bottom,
                            const vector<Blob*>& top) {
  const real_t* bottom_data = bottom[0]->gpu_data();
  real_t* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  TanHForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

}  // namespace caffe
