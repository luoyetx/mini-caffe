#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "./relu_layer.hpp"

namespace caffe {

__global__ void ReLUForward(const int n, const real_t* in, real_t* out,
    real_t negative_slope) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0 ? in[index] : in[index] * negative_slope;
  }
}

void ReLULayer::Forward_gpu(const vector<Blob*>& bottom,
                            const vector<Blob*>& top) {
  const real_t* bottom_data = bottom[0]->gpu_data();
  real_t* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  real_t negative_slope = this->layer_param_.relu_param().negative_slope();
  // NOLINT_NEXT_LINE(whitespace/operators)
  ReLUForward<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, negative_slope);
  CUDA_POST_KERNEL_CHECK;
}

}  // namespace caffe
